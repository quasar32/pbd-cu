
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdint.h>
#include <fcntl.h>
#include <errno.h>
#include <unistd.h>
#include <sys/stat.h>
#include <time.h>
#include <string.h>

#define FPS 60
#define N_BEADS 8 

#define DT (1.0f / FPS)
#define N_STEPS 100
#define SDT (DT / N_STEPS)
#define STS (N_STEPS * FPS)

struct bead {
  float radius;
  float mass;
  float2 pos;
  float2 prev_pos;
  float2 vel;
};

struct wire {
  float2 pos;
  float radius;
};

static bead(*host_groups)[N_BEADS]; 
static bead(*device_groups)[N_BEADS]; 
static wire host_wire = {{0.0f, 0.0f}, 0.8f};
static __constant__ __device__ wire device_wire = {{0.0f, 0.0f}, 0.8f};
static int n_groups = 1;
static int ends_only;
static FILE **csvs;

__constant__ __device__ float2 gravity = {0.0F, -10.0F};

__device__ float2 operator*(float2 a, float b) {
  return (float2) {a.x * b, a.y * b};
}

__device__ float2 operator-(float2 a, float2 b) {
  return (float2) {a.x - b.x, a.y - b.y};
}

__device__ float2 operator+(float2 a, float2 b) {
  return (float2) {a.x + b.x, a.y + b.y};
}

__device__ float dot(float2 a, float2 b) {
  return a.x * b.x + a.y * b.y; 
}

__device__ float length(float2 a) {
  return sqrtf(a.x * a.x + a.y * a.y);
}

__device__ float2 operator+=(float2 &a, float2 b) {
  a.x += b.x;
  a.y += b.y;
  return a; 
}

__device__ float2 operator/(float2 a, float b) {
  return (float2) {a.x / b, a.y / b};
}

__device__ void start_step(bead *a) {
  a->vel += gravity * SDT; 
  a->prev_pos = a->pos;
  a->pos += a->vel * SDT; 
}

__device__ void end_step(bead *a) {
  a->vel = a->pos - a->prev_pos;
  a->vel = a->vel * STS;
}

__device__ void bead_col(bead *a, bead *b) {
  float2 dir = b->pos - a->pos;
  float d = length(dir);
  if (d == 0.0f || d > a->radius + b->radius)
    return;
  dir = dir / d;
  float corr = (a->radius + b->radius - d) / 2.0f;
  a->pos = a->pos - dir * corr;
  b->pos = b->pos + dir * corr;
  float v0a = dot(a->vel, dir); 
  float v0b = dot(b->vel, dir); 
  float ma = a->mass;
  float mb = b->mass;
  float mt = ma + mb;
  float vc = ma * v0a + mb * v0b;
  float v1a = (vc - mb * (v0a - v0b)) / mt;
  float v1b = (vc - ma * (v0b - v0a)) / mt; 
  a->vel = a->vel + dir * (v1a - v0a); 
  b->vel = b->vel + dir * (v1b - v0b); 
}

__device__ void keep_on_wire(bead *a, wire *b) {
  float2 dir = a->pos - b->pos;
  float len = length(dir); 
  if (len == 0.0f)
    return;
  dir = dir / len;
  float lambda = b->radius - len;
  a->pos = a->pos + dir * lambda;
}

__global__ void update_sim(bead(*groups)[N_BEADS], int n_groups) {
  int group_idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (group_idx >= n_groups)
    return;
  bead *beads = groups[group_idx];
  for (int s = 0; s < N_STEPS; s++) {
    int i, j;
    for (i = 0; i < N_BEADS; i++)
      start_step(beads + i);
    for (i = 0; i < N_BEADS; i++)
      keep_on_wire(beads + i, &device_wire);
    for (i = 0; i < N_BEADS; i++)
      end_step(beads + i);
    for (i = 0; i < N_BEADS; i++) {
      for (j = 0; j < i; j++)
        bead_col(beads + i, beads + j);
    }
  }
}

static void die(const char *fn, int err) {
  fprintf(stderr, "%s(%d)\n", fn, err);
  exit(EXIT_FAILURE);
}

static void parse_args(int argc, char **argv) {
  int c;
  while ((c = getopt(argc, argv, "eg:")) != -1) {
    switch (c) {
    case 'e':
      /* output only first and last frame */
      ends_only = 1;
      break;
    case 'g':
      /* number of groups */
      if (sscanf(optarg, "%d\n", &n_groups) != 1) {
        fprintf(stderr, "group number is invalid\n");
        exit(1);
      }
      if (n_groups < 0) {
        fprintf(stderr, "negative groups\n");
        exit(1);
      }
      if (n_groups > 65536) {
        fprintf(stderr, "too many groups\n");
        exit(1);
      }
      break;
    case '?': 
      exit(1);
    }
  }
}

static void init_beads(void) {
  host_groups = new bead[n_groups][N_BEADS]; 
  for (int i = 0; i < n_groups; i++) { 
    float r = 0.1f;
    float rot = 0.0f;
    for (int j = 0; j < N_BEADS; j++) {
      bead *bd = &host_groups[i][j];
      bd->radius = r;
      bd->mass = (float) M_PI * r * r; 
      bd->pos.x = host_wire.pos.x + host_wire.radius * cosf(rot);
      bd->pos.y = host_wire.pos.y + host_wire.radius * sinf(rot);
      rot += (float) M_PI / N_BEADS;
      r = 0.05f + drand48() * 0.1f;
    }
  }
  hipError_t err = hipMalloc((void **) &device_groups, 
      n_groups * N_BEADS * sizeof(bead)); 
  if (err != hipSuccess)
    die("hipMalloc", err);
  err = hipMemcpy(device_groups, host_groups, 
      n_groups * N_BEADS * sizeof(bead), hipMemcpyHostToDevice);
  if (err != hipSuccess)
    die("hipMemcpy", err);
}

#if 0
static void update_sim(void) {
  cl_event ev;
  cl_int err = clEnqueueNDRangeKernel(cmdq, kernel, 1, NULL,
      (size_t[]) {n_groups}, NULL, 0, NULL, &ev);
  if (err != CL_SUCCESS)
    die("clEnqueueTask", err);
  err = clWaitForEvents(1, &ev);
  if (err != CL_SUCCESS)
    die("clWaitForEvents", err);
  cl_ulong start, end;
  err = clGetEventProfilingInfo(ev, CL_PROFILING_COMMAND_START, 8, &start, NULL);
  if (err != CL_SUCCESS)
    die("clGetEventProfilingInfo", err);
  err = clGetEventProfilingInfo(ev, CL_PROFILING_COMMAND_END, 8, &end, NULL);
  if (err != CL_SUCCESS)
    die("clGetEventProfilingInfo", err);
  elapsed += end - start;
  clReleaseEvent(ev);
  err = clEnqueueReadBuffer(cmdq, groups_mem, CL_TRUE, 0, 
      n_groups * sizeof(*host_groups), host_groups, 0, NULL, NULL);
  if (err != CL_SUCCESS)
    die("clEnqueueReadBuffer", err);
}
#endif

static FILE *open_csv(int i) {
  char buf[64]; 
  sprintf(buf, "out%03d.csv", i); 
  sprintf(buf, "out%06d.csv", i); 
  FILE *csv = fopen(buf, "wb");
  if (!csv)
    die("fopen", errno);
  return csv;
}

static void open_all(void) {
  csvs = new FILE*[n_groups];
  for (int i = 0; i < n_groups; i++) {
    char buf[64]; 
    sprintf(buf, "out%03d.csv", i); 
    csvs[i] = open_csv(i); 
  }
}

static void print_header(FILE *csv) {
  fprintf(csv, "f,t,x,y,r\n");
}

static void print_sim_one(FILE *csv, bead *host_groups, int frame) {
  for (int i = 0; i < N_BEADS; i++) {
    fprintf(csv, "%d,%d,%f,%f,%f\n", frame, 0, 
        host_groups[i].pos.x, host_groups[i].pos.y, host_groups[i].radius); 
  }
  fprintf(csv, "%d,%d,%f,%f,%f\n", frame, 1, 
      host_wire.pos.x, host_wire.pos.y, host_wire.radius); 
}

static void print_sim_all(int frame) {
  for (int i = 0; i < n_groups; i++) 
    print_sim_one(csvs[i], host_groups[i], frame);
}

static void update_sim_def(void) {
  int n_blocks = (n_groups + 255) / 256;
  update_sim<<<n_blocks, 256>>>(device_groups, n_groups);
  int err = hipGetLastError();
  if (err != hipSuccess) 
    die("update_sim", err);
}

static void copy_device_beads(bead(*host_groups)[N_BEADS]) {
    int err = hipMemcpy(host_groups, device_groups, 
        n_groups * N_BEADS * sizeof(bead), hipMemcpyDeviceToHost);
    if (err != hipSuccess) 
      die("hipMemcpy", err);
}

int main(int argc, char **argv) {
  parse_args(argc, argv);
  init_beads();
  if (ends_only) {
    int f;
    for (f = 0; f < 10 * FPS; f++) 
      update_sim_def();
    bead(*end)[N_BEADS] = new bead[n_groups][N_BEADS];
    copy_device_beads(end);
    for (int i = 0; i < n_groups; i++) {
      FILE *csv = open_csv(i);
      print_header(csv);
      print_sim_one(csv, host_groups[i], 0);
      print_sim_one(csv, end[i], 1);
      fclose(csv);
    }
    delete[] end;
  } else {
    open_all();
    for (int i = 0; i < n_groups; i++)
      print_header(csvs[i]);
    int f;
    for (f = 0; f < 10 * FPS; f++) {
      print_sim_all(f);
      update_sim_def();
      copy_device_beads(host_groups);
    }
    print_sim_all(f);
    for (int i = 0; i < n_groups; i++) 
      fclose(csvs[i]);
    delete[] csvs;
  }
  delete[] host_groups;
  hipFree(device_groups);
  return 0;
}
